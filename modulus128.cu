#include <stdint.h>
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>           // hipHostFree()
#include <CUDASieve/cudasieve.hpp>  // CudaSieve::getHostPrimes()                   

#include "cuda_uint128.h"

int main(int argc, char ** argv)
{
  uint128_t x = (uint128_t) 1 << 120;

  if(argc >= 2)
    x = string_to_u128((std::string)argv[3]);

  std::size_t st;
  uint32_t bottom = std::stoul(argv[1], &st, 10);
  uint32_t top = std::stoul(argv[2], &st, 10);

  size_t len;
  uint64_t * primes = CudaSieve::getHostPrimes(bottom, top, len);

  for(uint32_t i = 0; i < len; i++){
	uint64_t r;	
	uint128_t y = uint128_t::div128to128(x, primes[i], &r);
	if(r == 0) std::cout << primes[i] << std::endl;
  }

  hipHostFree(primes);

  std::cout << x << std::endl;

  return 0;

}
