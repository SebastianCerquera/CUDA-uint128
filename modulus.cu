#include "hip/hip_runtime.h"
#include <CUDASieve/host.hpp>

#include <iostream>
#include <stdio.h>
#include <math.h>
#include <cinttypes>
#include <vector>
#include <ctime>

int main(int argc, char* argv[])
{
  uint64_t * primes = NULL;
  uint64_t size = 1000;
  // start the timer
  clock_t start_time = clock();
  float elapsed_time;

  KernelData * kerneldata = new KernelData;
  kerneldata->allocate();

  hipHostMalloc(&primes, size*sizeof(uint64_t));

  uint64_t count = 0;
  count = kerneldata->getCount();
  std::cout << "Kernel Size : " << count << std::endl;

  hipHostFree(primes);
  hipFree(primes);

  kerneldata->deallocate();

  elapsed_time = (clock() - start_time)/((double) CLOCKS_PER_SEC);
  std::cout << "total time : " << elapsed_time << " seconds" << std::endl;

  hipDeviceReset();
  return 0;
}
